#include "hip/hip_runtime.h"
/* Host code for the Jacobi method of solving a system of linear equations 
 * by iteration.

 * Build as follws: make clean && make

 * Author: Naga Kandasamy
 * Date modified: May 21, 2020
*/

#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "jacobi_iteration.h"

/* Include the kernel code */
#include "jacobi_iteration_kernel.cu"

/* Uncomment the line below if you want the code to spit out debug information. */ 
/* #define DEBUG */

int main(int argc, char **argv) 
{
	if (argc > 1) {
		printf("This program accepts no arguments\n");
		exit(EXIT_FAILURE);
	}

    matrix_t  A;                    /* N x N constant matrix */
	matrix_t  B;                    /* N x 1 b matrix */
	matrix_t reference_x;           /* Reference solution */ 
	matrix_t gpu_naive_solution_x;  /* Solution computed by naive kernel */
    matrix_t gpu_opt_solution_x;    /* Solution computed by optimized kernel */

	/* Initialize the random number generator */
    srand(time(NULL));
    
    struct timeval start, stop;

	/* Generate diagonally dominant matrix */ 
    printf("\nGenerating %d x %d system\n", MATRIX_SIZE, MATRIX_SIZE);
	A = create_diagonally_dominant_matrix(MATRIX_SIZE, MATRIX_SIZE);
	if (A.elements == NULL) {
        printf("Error creating matrix\n");
        exit(EXIT_FAILURE);
	}
	
    /* Create the other vectors */
    B = allocate_matrix_on_host(MATRIX_SIZE, 1, 1);
	reference_x = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);
	gpu_naive_solution_x = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);
    gpu_opt_solution_x = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);

#ifdef DEBUG
	print_matrix(A);
	print_matrix(B);
	print_matrix(reference_x);
#endif

    /* Compute Jacobi solution on CPU */
    printf("\nPerforming Jacobi iteration on the CPU\n");
    gettimeofday (&start, NULL);
    compute_gold(A, reference_x, B);
    gettimeofday (&stop, NULL);
    printf ("Reference Execution Time = %fs\n", (float) (stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float) 1000000));
    display_jacobi_solution(A, reference_x, B); /* Display statistics */
	
	/* Compute Jacobi solution on device. Solutions are returned 
       in gpu_naive_solution_x and gpu_opt_solution_x. */
    printf("\nPerforming Jacobi iteration on device\n");
	compute_on_device(A, gpu_naive_solution_x, gpu_opt_solution_x, B);
    display_jacobi_solution(A, gpu_naive_solution_x, B); /* Display statistics */
    display_jacobi_solution(A, gpu_opt_solution_x, B); 
    
    free(A.elements); 
	free(B.elements); 
	free(reference_x.elements); 
	free(gpu_naive_solution_x.elements);
    free(gpu_opt_solution_x.elements);
	
    exit(EXIT_SUCCESS);
}


/* FIXME: Complete this function to perform Jacobi calculation on device */
void compute_on_device(const matrix_t A, matrix_t gpu_naive_sol_x, 
                       matrix_t gpu_opt_sol_x, const matrix_t B)
{
    struct timeval start, stop;
    int size = A.num_rows * A.num_columns;
    unsigned int num_rows = A.num_rows;
    unsigned int num_cols = A.num_columns;
    unsigned int done = 0;
    int num_iter_naive = 0;
    double ssd, mse, *ssd_dev;
    int pingpong = 1;

    /* Naive implementation */
    float *A_device = NULL;
    float *B_device = NULL;
    float *naive = NULL;
    float *opt = NULL;
    float *new_A = (float *)malloc(size * sizeof(float));
    int *mutex_on_device = NULL;

    /* Allocate n x 1 matrix to hold iteration values */
    float *new_x = NULL;

    /* Create grid and thread block sizes */
    dim3 threads(THREAD_BLOCK_SIZE, 1);
    dim3 grid(MATRIX_SIZE / THREAD_BLOCK_SIZE, 1);

    gettimeofday (&start, NULL);
    /* Malloc CUDA kernel arguments */
    hipMalloc((void **)&A_device, size * sizeof(float));
    hipMalloc((void **)&B_device, MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&naive, MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&opt, MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&new_x, MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&ssd_dev, sizeof(double));

    /* Copy from host to device */
    hipMemcpy(A_device, A.elements, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B.elements, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(naive, B.elements, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(opt, B.elements, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

    /* Allocate space for the lock on GPU and initialize it */
	hipMalloc((void **)&mutex_on_device, sizeof(int));
	hipMemset(mutex_on_device, 0, sizeof(int));

    while(!done)
    {
        (pingpong) ? (jacobi_iteration_kernel_naive<<< grid, threads >>>(A_device, naive, B_device, new_x, ssd_dev, mutex_on_device)) : \
                        (jacobi_iteration_kernel_naive<<< grid, threads >>>(A_device, new_x, B_device, naive, ssd_dev, mutex_on_device));
        hipDeviceSynchronize();
        
        num_iter_naive++;
        pingpong = !pingpong;
        hipMemcpy(&ssd, ssd_dev, sizeof(double), hipMemcpyDeviceToHost);
        mse = sqrt (ssd); /* Mean squared error. */
        // printf("Iteration: %d. MSE = %f\n", num_iter_naive, mse);
        if (mse <= THRESHOLD)
            done = 1;
    }

    (!pingpong) ? (hipMemcpy(gpu_naive_sol_x.elements, naive, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost)) : \
                        (hipMemcpy(gpu_naive_sol_x.elements, new_x, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost));

    gettimeofday (&stop, NULL);
    printf("\nConvergence achieved after %d iterations \n", num_iter_naive);
    printf ("CUDA (Naive) Execution Time = %fs\n\n", (float) (stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float) 1000000));

    /* Optimized implementation */
    done = 0;
    pingpong = 1;
    num_iter_naive = 0;

    gettimeofday (&start, NULL);
    
    /* Convert A matrix from row major format to column major format */
    for (int i = 0; i < num_cols; ++i)
        for (int j = 0; j < num_rows; ++j)
            new_A[ i * num_rows + j ] = A.elements[ j * num_cols + i ];

    /* Malloc CUDA kernel arguments */
    hipMalloc((void **)&A_device, size * sizeof(float));
    hipMalloc((void **)&B_device, MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&naive, MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&opt, MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&new_x, MATRIX_SIZE * sizeof(float));
    hipMalloc((void **)&ssd_dev, sizeof(double));

    /* Allocate space for the lock on GPU and initialize it */
    hipMemcpy(A_device, new_A, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_device, B.elements, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(naive, B.elements, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(opt, B.elements, MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

    /* Allocate space for the lock on GPU and initialize it */
	hipMalloc((void **)&mutex_on_device, sizeof(int));
	hipMemset(mutex_on_device, 0, sizeof(int));
    
    while(!done)
    {
        (pingpong) ? (jacobi_iteration_kernel_optimized<<< grid, threads >>>(A_device, opt, B_device, new_x, ssd_dev, mutex_on_device)) : \
                        (jacobi_iteration_kernel_optimized<<< grid, threads >>>(A_device, new_x, B_device, opt, ssd_dev, mutex_on_device));
        hipDeviceSynchronize();
        
        num_iter_naive++;
        pingpong = !pingpong;
        hipMemcpy(&ssd, ssd_dev, sizeof(double), hipMemcpyDeviceToHost);
        mse = sqrt (ssd); /* Mean squared error. */
        // printf("Iteration: %d. MSE = %f\n", num_iter_naive, mse);
        if (mse <= THRESHOLD)
            done = 1;
    }

    (!pingpong) ? (hipMemcpy(gpu_opt_sol_x.elements, opt, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost)) : \
                        (hipMemcpy(gpu_opt_sol_x.elements, new_x, MATRIX_SIZE * sizeof(float), hipMemcpyDeviceToHost));

    gettimeofday (&stop, NULL);
    printf("\nConvergence achieved after %d iterations \n", num_iter_naive);
    printf ("CUDA (Optimized) Execution Time = %fs\n\n", (float) (stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float) 1000000));
    
    hipFree(A_device);
    hipFree(B_device);
    hipFree(naive);
    hipFree(opt);
    hipFree(new_x);
    free(new_A);

    return;
}

/* Allocate matrix on the device of same size as M */
matrix_t allocate_matrix_on_device(const matrix_t M)
{
    matrix_t Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void **)&Mdevice.elements, size);
    return Mdevice;
}

/* Allocate a matrix of dimensions height * width.
   If init == 0, initialize to all zeroes.  
   If init == 1, perform random initialization.
*/
matrix_t allocate_matrix_on_host(int num_rows, int num_columns, int init)
{	
    matrix_t M;
    M.num_columns = num_columns;
    M.num_rows = num_rows;
    int size = M.num_rows * M.num_columns;
		
	M.elements = (float *)malloc(size * sizeof(float));
	for (unsigned int i = 0; i < size; i++) {
		if (init == 0) 
            M.elements[i] = 0; 
		else
            M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
	}
    
    return M;
}	

/* Copy matrix to device */
void copy_matrix_to_device(matrix_t Mdevice, const matrix_t Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
    return;
}

/* Copy matrix from device to host */
void copy_matrix_from_device(matrix_t Mhost, const matrix_t Mdevice)
{
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
    return;
}

/* Prints the matrix out to screen */
void print_matrix(const matrix_t M)
{
	for (unsigned int i = 0; i < M.num_rows; i++) {
        for (unsigned int j = 0; j < M.num_columns; j++) {
			printf("%f ", M.elements[i * M.num_columns + j]);
        }
		
        printf("\n");
	} 
	
    printf("\n");
    return;
}

/* Returns a floating-point value between [min, max] */
float get_random_number(int min, int max)
{
    float r = rand()/(float)RAND_MAX;
	return (float)floor((double)(min + (max - min + 1) * r));
}

/* Check for errors in kernel execution */
void check_CUDA_error(const char *msg)
{
	hipError_t err = hipGetLastError();
	if ( hipSuccess != err) {
		printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}	
    
    return;    
}

/* Create diagonally dominant matrix */
matrix_t create_diagonally_dominant_matrix(unsigned int num_rows, unsigned int num_columns)
{
	matrix_t M;
	M.num_columns = num_columns;
	M.num_rows = num_rows; 
	unsigned int size = M.num_rows * M.num_columns;
	M.elements = (float *)malloc(size * sizeof(float));
    if (M.elements == NULL)
        return M;

	/* Create a matrix with random numbers between [-.5 and .5] */
    unsigned int i, j;
	for (i = 0; i < size; i++)
        M.elements[i] = get_random_number (MIN_NUMBER, MAX_NUMBER);
	
	/* Make diagonal entries large with respect to the entries on each row. */
	for (i = 0; i < num_rows; i++) {
		float row_sum = 0.0;		
		for (j = 0; j < num_columns; j++) {
			row_sum += fabs(M.elements[i * M.num_rows + j]);
		}
		
        M.elements[i * M.num_rows + i] = 0.5 + row_sum;
	}

    return M;
}

