#include "hip/hip_runtime.h"
/* FIXME: Edit this file to complete the functionality of 2D separable 
 * convolution on the GPU. You may add additional kernel functions 
 * as necessary. 
 */

__global__ void convolve_rows_kernel_naive(float *result, float *input, float *kernel, int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;

    x = blockDim.x * blockIdx.x + threadIdx.x;
    y = blockDim.y * blockIdx.y + threadIdx.y;

    j1 = x - half_width;
    j2 = x + half_width;

    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_cols) 
        j2 = num_cols - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - x;

    j1 = j1 - x + half_width; /* Obtain operating width of the kernel */
    j2 = j2 - x + half_width;

    /* Convolve along row */
    result[y * num_cols + x] = 0.0f;
    for(i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += kernel[j] * input[y * num_cols + x + i];

    return;
}

__global__ void convolve_columns_kernel_naive(float *result, float *input, float *kernel, int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;

    x = blockDim.x * blockIdx.x + threadIdx.x;
    y = blockDim.y * blockIdx.y + threadIdx.y;

    j1 = y - half_width;
    j2 = y + half_width;

    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_rows) 
        j2 = num_rows - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - y;

    j1 = j1 - y + half_width; /* Obtain the operating width of the kernel.*/
    j2 = j2 - y + half_width;

    /* Convolve along column */
    result[y * num_cols + x] = 0.0f;
    for (i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += kernel[j] * input[y * num_cols + x + (i * num_cols)];

    return;
}

__global__ void convolve_rows_kernel_optimized(float *result, float *input, int num_cols, int num_rows, int half_width)
{
    __shared__ float input_ts[(THREAD_BLOCK_SIZE + HALF_WIDTH * 2) * THREAD_BLOCK_SIZE];
    int i, i1;
    int j, j1, j2;
    int x, y;

    x = blockDim.x * blockIdx.x + threadIdx.x;
    y = blockDim.y * blockIdx.y + threadIdx.y;

    /* Load the left halo elements from the previous tile. 
     * The number of halo elements will be half_width.
     */
    int left_halo_index = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x >= (blockDim.x - half_width)) {
        if (left_halo_index < 0)
            input_ts[(threadIdx.x - (blockDim.x - half_width)) + threadIdx.y * (THREAD_BLOCK_SIZE + HALF_WIDTH * 2)] = 0.0;
        else
            input_ts[(threadIdx.x - (blockDim.x - half_width)) + threadIdx.y * (THREAD_BLOCK_SIZE + HALF_WIDTH * 2)] = input[left_halo_index + y * num_rows];
    }

    /* Load the center elements for the tile */
    if (x < num_cols)
        input_ts[half_width + threadIdx.x + threadIdx.y * (THREAD_BLOCK_SIZE + HALF_WIDTH * 2)] = input[y * num_rows + x];
    else
        input_ts[half_width + threadIdx.x + threadIdx.y * (THREAD_BLOCK_SIZE + HALF_WIDTH * 2)] = 0.0;

    /* Load the right halo elements from the next tile.
     * The number of halo elements will again be half_width.
     */
    int right_halo_index = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x < half_width) {
        if (right_halo_index >= num_cols)
            input_ts[threadIdx.x + (blockDim.x + half_width) + threadIdx.y * (THREAD_BLOCK_SIZE + HALF_WIDTH * 2)] = 0.0;
        else
            input_ts[threadIdx.x + (blockDim.x + half_width) + threadIdx.y * (THREAD_BLOCK_SIZE + HALF_WIDTH * 2)] = input[right_halo_index + y * num_rows];
    }

    __syncthreads();

    j1 = x - half_width;
    j2 = x + half_width;

    /* Clamp at the edges of the matrix */
    if (j1 < 0)
        j1 = 0;
    if (j2 >= num_cols)
        j2 = num_cols - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - x;

    j1 = j1 - x + half_width; /* Obtain operating width of the kernel */
    j2 = j2 - x + half_width;

    /* Convolve along row */
    result[y * num_cols + x] = 0.0f;
    for(i = i1, j = j1; j <= j2; j++, i++)
        // result[y * num_cols + x] += kernel_c[j] * input[y * num_cols + x + i];
        result[y * num_cols + x] += kernel_c[j] * input_ts[threadIdx.y * blockDim.y + threadIdx.x + i + half_width];
    
    return;
}

__global__ void convolve_columns_kernel_optimized(float *result, float *input, int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;

    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;

    j1 = y - half_width;
    j2 = y + half_width;
    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_rows) 
        j2 = num_rows - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - y; 
    
    j1 = j1 - y + half_width; /* Obtain the operating width of the kernel.*/
    j2 = j2 - y + half_width;

    /* Convolve along column */            
    result[y * num_cols + x] = 0.0f;
    for (i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += kernel_c[j] * input[y * num_cols + x + (i * num_cols)];
    
    return;
}
 