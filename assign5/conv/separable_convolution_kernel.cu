#include "hip/hip_runtime.h"
/* FIXME: Edit this file to complete the functionality of 2D separable 
 * convolution on the GPU. You may add additional kernel functions 
 * as necessary. 
 */

__global__ void convolve_rows_kernel_naive(float *result, float *input, float *kernel, int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;

    x = blockDim.x * blockIdx.x + threadIdx.x;
    y = blockDim.y * blockIdx.y + threadIdx.y;

    j1 = x - half_width;
    j2 = x + half_width;

    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_cols) 
        j2 = num_cols - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - x;

    j1 = j1 - x + half_width; /* Obtain operating width of the kernel */
    j2 = j2 - x + half_width;

    /* Convolve along row */
    result[y * num_cols + x] = 0.0f;
    for(i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += kernel[j] * input[y * num_cols + x + i];

    return;
}

__global__ void convolve_columns_kernel_naive(float *result, float *input, float *kernel, int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;

    x = blockDim.x * blockIdx.x + threadIdx.x;
    y = blockDim.y * blockIdx.y + threadIdx.y;

    j1 = y - half_width;
    j2 = y + half_width;

    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_rows) 
        j2 = num_rows - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - y;

    j1 = j1 - y + half_width; /* Obtain the operating width of the kernel.*/
    j2 = j2 - y + half_width;

    /* Convolve along column */
    result[y * num_cols + x] = 0.0f;
    for (i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += kernel[j] * input[y * num_cols + x + (i * num_cols)];

    return;
}

__global__ void convolve_rows_kernel_optimized(float *result, float *input, int num_cols, int num_rows, int half_width)
{
    __shared__ float input_ts[(THREAD_BLOCK_SIZE + HALF_WIDTH * 2) * THREAD_BLOCK_SIZE];
    int i, i1;
    int j, j1, j2;
    int x, y;

    x = blockDim.x * blockIdx.x + threadIdx.x;
    y = blockDim.y * blockIdx.y + threadIdx.y;

    /* Load the left halo elements from the previous tile. 
     * The number of halo elements will be half_width.
     */
    int left_halo_index = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x >= (blockDim.x - half_width)) {
        if (left_halo_index < 0)
            input_ts[threadIdx.y * blockDim.y + (threadIdx.x - (blockDim.x - half_width))] = 0.0;
        else
            input_ts[threadIdx.y * blockDim.y + (threadIdx.x - (blockDim.x - half_width))] = input[left_halo_index + y * num_rows];
    }

    /* Load the center elements for the tile */
    if (x < num_cols)
        input_ts[threadIdx.y * blockDim.y + (threadIdx.x + half_width)] = input[y * num_rows + x];
    else
        input_ts[threadIdx.y * blockDim.y + (threadIdx.x + half_width)] = 0.0;

    /* Load the right halo elements from the next tile.
     * The number of halo elements will again be half_width.
     */
    int right_halo_index = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x < half_width) {
        if (right_halo_index >= num_cols)
            input_ts[threadIdx.y * blockDim.y + threadIdx.x + (blockDim.x + half_width)] = 0.0;
        else
            input_ts[threadIdx.y * blockDim.y + threadIdx.x + (blockDim.x + half_width)] = input[right_halo_index + y * num_rows];
    }

    __syncthreads();

    j1 = x - half_width;
    j2 = x + half_width;

    /* Clamp at the edges of the matrix */
    if (j1 < 0)
        j1 = 0;
    if (j2 >= num_cols)
        j2 = num_cols - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - x;

    j1 = j1 - x + half_width; /* Obtain operating width of the kernel */
    j2 = j2 - x + half_width;

    /* Convolve along row */
    result[y * num_cols + x] = 0.0f;
    for (i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += kernel_c[j] * input_ts[threadIdx.y * blockDim.x + (threadIdx.x + half_width) + i];
    
    return;
}

__global__ void convolve_columns_kernel_optimized(float *result, float *input, int num_cols, int num_rows, int half_width)
{
    __shared__ float input_ts[(THREAD_BLOCK_SIZE + HALF_WIDTH * 2) * THREAD_BLOCK_SIZE];
    int i, i1;
    int j, j1, j2;
    int x, y;

    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;

    /* Load the left halo elements from the previous tile.
     * The number of halo elements will be half_width.
     */
    int left_halo_index = (blockIdx.y - 1) * blockDim.y + threadIdx.y;
    if (threadIdx.y >= (blockDim.y - half_width)) {
        if (left_halo_index < 0)
            input_ts[(threadIdx.y - (blockDim.y - half_width)) * blockDim.y + threadIdx.x] = 0.0;
        else
            input_ts[(threadIdx.y - (blockDim.y - half_width)) * blockDim.y + threadIdx.x] = input[left_halo_index * num_rows + x];
    }

    /* Load the center elements for the tile */
    if (y < num_rows)
        input_ts[(threadIdx.y + half_width) * blockDim.y + threadIdx.x] = input[y * num_rows + x];
    else
        input_ts[(threadIdx.y + half_width) * blockDim.y + threadIdx.x] = 0.0;

    /* Load the right halo elements from the next tile.
     * The number of halo elements will again be half_width.
     */
    int right_halo_index = (blockIdx.y + 1) * blockDim.y + threadIdx.y;
    if (threadIdx.y < half_width) {
        if (right_halo_index >= num_rows)
            input_ts[(threadIdx.y + (blockDim.y + half_width)) * blockDim.y + threadIdx.x] = 0.0;
        else
            input_ts[(threadIdx.y + (blockDim.y + half_width)) * blockDim.y + threadIdx.x] = input[right_halo_index * num_rows + x];
    }
 
    __syncthreads();

    j1 = y - half_width;
    j2 = y + half_width;
    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_rows) 
        j2 = num_rows - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - y; 
    
    j1 = j1 - y + half_width; /* Obtain the operating width of the kernel.*/
    j2 = j2 - y + half_width;

    /* Convolve along column */
    result[y * num_cols + x] = 0.0f;
    for (i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += kernel_c[j] * input_ts[(threadIdx.y + half_width) * blockDim.x + threadIdx.x + (i * blockDim.x)];
    
    return;
}