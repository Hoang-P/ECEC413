#include "hip/hip_runtime.h"
/* Blur filter. Device code. */

#ifndef _BLUR_FILTER_KERNEL_H_
#define _BLUR_FILTER_KERNEL_H_

#include "blur_filter.h"

__global__ void 
blur_filter_kernel (const float *in, float *out, int size)
{
    /* Obtain thread location within the block */
    int tx, ty;
    tx = threadIdx.x;
    ty = threadIdx.y;

    int bx, by;
    bx = blockIdx.x;
    by = blockIdx.y;

    int bdimx, bdimy;
    bdimx = blockDim.x;
    bdimy = blockDim.y;
    
    // int imgSize = size;
    int row = bdimy * by + ty;
    int col = bdimx * bx + tx;

    int i, j;
    int curr_row, curr_col;
    float blur_value;
    int num_neighbors;

    /* Apply blur filter to current pixel */
    blur_value = 0.0;
    num_neighbors = 0;
    for (i = -BLUR_SIZE; i < (BLUR_SIZE + 1); i++) {
        for (j = -BLUR_SIZE; j < (BLUR_SIZE + 1); j++) {
            /* Accumulate values of neighbors while checking for 
             * boundary conditions */
            curr_row = row + i;
            curr_col = col + j;
            if ((curr_row > -1) && (curr_row < size) &&\
                    (curr_col > -1) && (curr_col < size)) {
                blur_value += in[curr_row * size + curr_col];
                num_neighbors += 1;
            }
        }
    }
    out[row * size + col] = blur_value/num_neighbors;

    return;
}

#endif /* _BLUR_FILTER_KERNEL_H_ */
